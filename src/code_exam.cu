#include <stdio.h>
#include <hip/hip_runtime.h>

#define DRAW_START 128
#define DRAW_END 1024

// CUDA kernel to fill matrix (row-major order)
__global__ void fill_matrix_kernel(int* matrix, int rows, int cols, int value) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > DRAW_START && row < DRAW_END && col < DRAW_END && col > DRAW_START) {
        int idx = row * cols + col;
        matrix[idx] = value;
    }
}

void print_matrix(int *s, int x_len, int y_len) {
  for (int i = 0; i < y_len; i++) {
    for (int j = 0; j < x_len; j++) {
      printf("%d, ", s[i * x_len + j]);
    }
    printf("\n");
  }
}

void initialize_matrix(int *matrix, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      matrix[i * rows + j] = 0;
    }
  }
}

int main() {
    const int rows = 32768;
    const int cols = 32768;
    const size_t size = rows * cols * sizeof(int);
    
    // Host memory
    int* h_matrix = (int *)malloc(size);
    // initialize_matrix(h_matrix, rows, cols);
    
    // Device memory
    int* d_matrix;
    hipMalloc((void **)&d_matrix, size);

    // cudaMemcpy(d_matrix, h_matrix, size, cudaMemcpyHostToDevice);
    
    // Define grid and block dimensions
    dim3 block(32, 32);  // 256 threads per block
    dim3 grid(
        (cols + block.x - 1) / block.x,  // ceil(cols/block.x)
        (rows + block.y - 1) / block.y   // ceil(rows/block.y)
    );
    
    // Launch kernel
    fill_matrix_kernel<<<grid, block>>>(d_matrix, rows, cols, 9);
    // cudaDeviceSynchronize();
    
    // Copy result back to host
    // cudaMemcpy(h_matrix, d_matrix, size, cudaMemcpyDeviceToHost);
    
    // Verify values
    // print_matrix(h_matrix, rows, cols);
    
    // Cleanup
    free(h_matrix);
    hipFree(d_matrix);
    
    return 0;
}
